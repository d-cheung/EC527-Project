#include "hip/hip_runtime.h"
#include "IntegralImage.h"
#include "bitmap_image.hpp"
#include <cmath>

const float IntegralImage::cR = (float).2989;
const float IntegralImage::cG = (float).5870;
const float IntegralImage::cB = (float).1140;

float IntegralImage::getValue(int y, int x)
{
	return this->Matrix[y][x];
}

void IntegralImage::setValue(int y, int x, float value)
{ 
	this->Matrix[y][x] = value;
}

IntegralImage::IntegralImage()
{
	this->Width = 0;
	this->Height = 0;
	this->Matrix = NULL;
}

IntegralImage::IntegralImage(int width, int height)
{
	this->Width = width;
	this->Height = height;

	hipMalloc((void **) &(this->Matrix), height*sizeof(float *));
	float ** temp[height];
	hipMemcpy(temp, this->Matrix, height*sizeof(float *), hipMemcpyDeviceToHost);
	for (int ii = 0; ii < height; ii++)
		hipMalloc((void **) &(temp[ii]), (width)*sizeof(float));
	hipMemcpy(this->Matrix, temp, height*sizeof(float *), hipMemcpyHostToDevice);

}

IntegralImage::~IntegralImage()
{

	if (Matrix != NULL)
	{
		float ** temp[this->Height];
		hipMemcpy(temp, this->Matrix, (this->Height)*sizeof(float *), hipMemcpyDeviceToHost);

		for (int ii = 0; ii < this->Height; ii++)
			hipFree(temp[ii]);

		hipFree(this->Matrix);
	}

}

/* example kernel */
__global__ void kernelFromImageCols(unsigned char * d_image, float ** d_pic, int Width, int Height, unsigned int row_increment, unsigned int bytes_per_pixel)
{

	float colsum = (float)(0.0);
	unsigned char red, green, blue;
  	unsigned int  row_increment_ = row_increment;
	unsigned int  bytes_per_pixel_ = bytes_per_pixel;
	int width = Width;
	int height = Height;
	const float cR = (float).2989;
	const float cG = (float).5870;
	const float cB = (float).1140;

	unsigned int x = threadIdx.x + blockDim.x*blockIdx.x;
	if (x >= width) return;

	for (unsigned int y = 0; y < height; y++)
	{
		blue  = d_image[(y * row_increment_) + (x * bytes_per_pixel_ + 0)];
		green = d_image[(y * row_increment_) + (x * bytes_per_pixel_ + 1)];
		red   = d_image[(y * row_increment_) + (x * bytes_per_pixel_ + 2)];

		colsum += (cR * red + cG * green + cB * blue) / (float)255;
		d_pic[y][x] = colsum;
	}
  
}

__global__ void kernelFromImageRows(float ** d_pic, int Width, int Height){

	int width = Width;
	int height = Height;

	int y = threadIdx.x + blockDim.x*blockIdx.x;
	if (y >= height) return;

	float rowsum = d_pic[y][0];

	for (unsigned int x = 1; x < width; x++)
	{
		rowsum += d_pic[y][x];
		d_pic[y][x] = rowsum;
	}
}

IntegralImage * IntegralImage::FromImage(bitmap_image &h_image)
{
	int ThreadsPerBlock = 256;
	int BlocksPerGrid;

	IntegralImage * h_pic = new IntegralImage(h_image.width(), h_image.height());

	unsigned char * d_image;
	hipMalloc((void**) &d_image, (h_image.length_)*sizeof(unsigned char));
	hipMemcpy(d_image, h_image.data_, (h_image.length_)*sizeof(unsigned char), hipMemcpyHostToDevice);

	BlocksPerGrid = ((h_pic->Width / ThreadsPerBlock) + 1);
	kernelFromImageCols <<< BlocksPerGrid, ThreadsPerBlock >>> (d_image, h_pic->Matrix, h_pic->Width, h_pic->Height, h_image.row_increment_, h_image.bytes_per_pixel_);
 	hipDeviceSynchronize();

	BlocksPerGrid = ((h_pic->Height / ThreadsPerBlock) + 1);
	kernelFromImageRows <<< BlocksPerGrid, ThreadsPerBlock >>> (h_pic->Matrix, h_pic->Width, h_pic->Height);
 	hipDeviceSynchronize();

	hipFree(d_image);
	return h_pic; 
}



float IntegralImage::BoxIntegral(int row, int col, int rows, int cols)
{
	// The subtraction by one for row/col is because row/col is inclusive.
	int r1 = std::min(row, Height) - 1;
	int c1 = std::min(col, Width) - 1;
	int r2 = std::min(row + rows, Height) - 1;
	int c2 = std::min(col + cols, Width) - 1;

	float A = 0, B = 0, C = 0, D = 0;

	if (r1 >= 0 && c1 >= 0) A = Matrix[r1][c1];
	if (r1 >= 0 && c2 >= 0) B = Matrix[r1][c2];
	if (r2 >= 0 && c1 >= 0) C = Matrix[r2][c1];
	if (r2 >= 0 && c2 >= 0) D = Matrix[r2][c2];

	return std::max((float)0, A - B - C + D);
}


// Get Haar Wavelet X repsonse
float IntegralImage::HaarX(int row, int column, int size)
{
	return BoxIntegral(row - size / 2, column, size, size / 2)
	 - 1 * BoxIntegral(row - size / 2, column - size / 2, size, size / 2);
}

// Get Haar Wavelet Y repsonse
float IntegralImage::HaarY(int row, int column, int size)
{
	return BoxIntegral(row, column - size / 2, size / 2, size)
	 - 1 * BoxIntegral(row - size / 2, column - size / 2, size / 2, size);
}


