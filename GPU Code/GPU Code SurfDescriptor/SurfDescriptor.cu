#include "hip/hip_runtime.h"
#include <vector>
#define _USE_MATH_DEFINES
#include <cmath>
#include "SurfDescriptor.h"
#include "IPoint.h"
#include "IntegralImage.h"
#include "FastHessian.h"

float SurfDescriptor::gauss25[49] = {
	(float)0.02350693969273,(float)0.01849121369071,(float)0.01239503121241,(float)0.00708015417522,(float)0.00344628101733,(float)0.00142945847484,(float)0.00050524879060, 
	(float)0.02169964028389,(float)0.01706954162243,(float)0.01144205592615,(float)0.00653580605408,(float)0.00318131834134,(float)0.00131955648461,(float)0.00046640341759, 
	(float)0.01706954162243,(float)0.01342737701584,(float)0.00900063997939,(float)0.00514124713667,(float)0.00250251364222,(float)0.00103799989504,(float)0.00036688592278,
	(float)0.01144205592615,(float)0.00900063997939,(float)0.00603330940534,(float)0.00344628101733,(float)0.00167748505986,(float)0.00069579213743,(float)0.00024593098864,
	(float)0.00653580605408,(float)0.00514124713667,(float)0.00344628101733,(float)0.00196854695367,(float)0.00095819467066,(float)0.00039744277546,(float)0.00014047800980,
	(float)0.00318131834134,(float)0.00250251364222,(float)0.00167748505986,(float)0.00095819467066,(float)0.00046640341759,(float)0.00019345616757,(float)0.00006837798818,
	(float)0.00131955648461,(float)0.00103799989504,(float)0.00069579213743,(float)0.00039744277546,(float)0.00019345616757,(float)0.00008024231247,(float)0.00002836202103
};


/// <summary>
/// Static one-call do it all function
/// </summary>
/// <param name="img"></param>
/// <param name="ipts"></param>
/// <param name="extended"></param>
/// <param name="upright"></param>
void SurfDescriptor::DecribeInterestPoints(std::vector<IPoint>* ipts, bool upright, bool extended, IntegralImage * img)
{
    SurfDescriptor des;
    des.DescribeInterestPoints(ipts, upright, extended, img);
}


/// <summary>
    /// Build descriptor vector for each interest point in the supplied list
    /// </summary>
    /// <param name="img"></param>
    /// <param name="ipts"></param>
    /// <param name="upright"></param>
void SurfDescriptor::DescribeInterestPoints(std::vector<IPoint>* ipts, bool upright, bool extended, IntegralImage *img)
{
	if (ipts->size() == 0) return;
	this->img = img;

	float * d_resX, * d_resY, * d_Ang;
	float * d_descriptor, * d_len;

	hipMalloc((void**) &d_resX,      (109)*sizeof(float));
	hipMalloc((void**) &d_resY,      (109)*sizeof(float));
	hipMalloc((void**) &d_Ang,       (109)*sizeof(float));
	hipMalloc((void**) &d_gauss25,    (49)*sizeof(float));
	hipMalloc((void**) &d_descriptor, (64)*sizeof(float));
	hipMalloc((void**) &d_len,        (16)*sizeof(float));

	hipMemcpy(d_gauss25, gauss25, 49*sizeof(float), hipMemcpyHostToDevice);

	for (std::vector<IPoint>::iterator ip = ipts->begin(); ip != ipts->end(); ++ip)
	{
		// determine descriptor size
		if (extended) ip->descriptorLength = 128;
		else ip->descriptorLength = 64;

		// if we want rotation invariance get the orientation
		if (!upright) GetOrientation(*ip, d_resX, d_resY, d_Ang);

		// Extract SURF descriptor
		GetDescriptor(*ip, upright, extended, d_descriptor, d_len);
	}

	hipFree(d_resX);
	hipFree(d_resY);
	hipFree(d_Ang);
	hipFree(d_gauss25);
	hipFree(d_len);
	hipFree(d_descriptor);
}
/*
__device__ double d_GetAngle(float X, float Y)
{
	if (X >= 0 && Y >= 0)
		return atan(Y / X);
	else if (X < 0 && Y >= 0)
		return M_PI - atan(-Y / X);
	else if (X < 0 && Y < 0)
		return M_PI + atan(Y / X);
 	else if (X >= 0 && Y < 0)
		return 2 * M_PI - atan(-Y / X);
	return 0;
}
*/

__global__ void cudaHaar6x6(float ** img, float * resX, float * resY, float * Ang, float * gauss25, int X, int Y, int S, int Height, int Width)
{
	int id[11] = {5, 4, 3, 2, 1, 0, 1, 2, 3, 4, 5};
	int height = Height;
	int width = Width;
	
	int i = id[threadIdx.x];
	int j = id[threadIdx.y];

	int idx;
	switch (threadIdx.x)
	{
		case 0:
			switch (threadIdx.y) {
				case 0:
				case 1:
				case 9:
				case 10:
					return;
				default:
					idx = threadIdx.y - 2;
			}
			break;
		case 1:
			switch (threadIdx.y) {
				case 0:
				case 10:
					return;
				default:
					idx = threadIdx.y + 6;
			}
			break;
		case 9:
			switch (threadIdx.y) {
				case 0:
				case 10:
					return;
				default:
					idx = threadIdx.y + 92;
			}
			break;
		case 10:
			switch (threadIdx.y) {
				case 0:
				case 1:
				case 9:
				case 10:
					return;
				default:
					idx = threadIdx.y + 101;
			}
			break;
		default:
			idx = threadIdx.x * 11 + threadIdx.y - 6;
	}

	float gauss = gauss25[id[i]*7 + id[j]];
	resX[idx] = gauss * d_HaarX(img, Y + j * S, X + i * S, 4 * S, height, width);
	resY[idx] = gauss * d_HaarY(img, Y + j * S, X + i * S, 4 * S, height, width);
//	Ang[idx] = (float)d_GetAngle(resX[idx], resY[idx]);
}

__global__ void cudaHaar6x6(float * img, float * resX, float * resY, float * Ang, float * gauss25, int X, int Y, int S, size_t Pitch, int Height, int Width)
{
	int id[11] = {5, 4, 3, 2, 1, 0, 1, 2, 3, 4, 5};
	int pitch = Pitch;
	int height = Height;
	int width = Width;
	
	int i = id[threadIdx.x];
	int j = id[threadIdx.y];

	int idx;
	switch (threadIdx.x)
	{
		case 0:
			switch (threadIdx.y) {
				case 0:
				case 1:
				case 9:
				case 10:
					return;
				default:
					idx = threadIdx.y - 2;
			}
			break;
		case 1:
			switch (threadIdx.y) {
				case 0:
				case 10:
					return;
				default:
					idx = threadIdx.y + 6;
			}
			break;
		case 9:
			switch (threadIdx.y) {
				case 0:
				case 10:
					return;
				default:
					idx = threadIdx.y + 92;
			}
			break;
		case 10:
			switch (threadIdx.y) {
				case 0:
				case 1:
				case 9:
				case 10:
					return;
				default:
					idx = threadIdx.y + 101;
			}
			break;
		default:
			idx = threadIdx.x * 11 + threadIdx.y - 6;
	}

	float gauss = gauss25[id[i]*7 + id[j]];
	resX[idx] = gauss * d_HaarX(img, Y + j * S, X + i * S, 4 * S, pitch, height, width);
	resY[idx] = gauss * d_HaarY(img, Y + j * S, X + i * S, 4 * S, pitch, height, width);
//	Ang[idx] = (float)d_GetAngle(resX[idx], resY[idx]);
  }

/// <summary>
    /// Determine dominant orientation for InterestPoint
    /// </summary>
    /// <param name="ip"></param>
void SurfDescriptor::GetOrientation(IPoint &ip, float * d_resX, float * d_resY, float * d_Ang)
{
  const unsigned char Responses = 109;
  float resX[Responses];
  float resY[Responses];
  float Ang[Responses];

  // Get rounded InterestPoint data
  int X = (int)floor(ip.x + (float)0.5);
  int Y = (int)floor(ip.y + (float)0.5);
  int S = (int)floor(ip.scale + (float)0.5);

  dim3 dimBlock(11, 11);


#ifdef FLOAT_SINGLE
  cudaHaar6x6 <<<1, dimBlock>>> (img->Matrix, d_resX, d_resY, d_Ang, d_gauss25, X, Y, S, img->Pitch, img->Height, img->Width);
#endif

#ifdef FLOAT_DOUBLE
  cudaHaar6x6 <<<1, dimBlock>>> (img->Matrix, d_resX, d_resY, d_Ang, d_gauss25, X, Y, S, img->Height, img->Width);
#endif

  hipDeviceSynchronize();

  hipMemcpy(resX, d_resX, (Responses)*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(resY, d_resY, (Responses)*sizeof(float), hipMemcpyDeviceToHost);
//  hipMemcpy( Ang, d_Ang,  (Responses)*sizeof(float), hipMemcpyDeviceToHost);

  for (unsigned char ii = 0; ii < Responses; ii++)
	Ang[ii] = (float)GetAngle(resX[ii], resY[ii]);

  // calculate the dominant direction 
  float sumX, sumY, max = 0, orientation = 0;
  float ang1, ang2;
  float pi = (float)M_PI;

  // loop slides pi/3 window around feature point
  for (ang1 = 0; ang1 < 2 * pi; ang1 += (float)0.15)
  {
	ang2 = (ang1 + pi / (float)3 > 2 * pi ? ang1 - 5 * pi / (float)3 : ang1 + pi / (float)3);
	sumX = sumY = 0;

	for (int k = 0; k < Responses; ++k)
	{
	  // determine whether the point is within the window
	  if (ang1 < ang2 && ang1 < Ang[k] && Ang[k] < ang2)
	  {
		sumX += resX[k];
		sumY += resY[k];
	  }
	  else if (ang2 < ang1 &&
		((Ang[k] > 0 && Ang[k] < ang2) || (Ang[k] > ang1 && Ang[k] < pi)))
	  {
		sumX += resX[k];
		sumY += resY[k];
	  }
	}

	// if the vector produced from this window is longer than all 
	// previous vectors then this forms the new dominant direction
	if (sumX * sumX + sumY * sumY > max)
	{
	  // store largest orientation
	  max = sumX * sumX + sumY * sumY;
	  orientation = (float)GetAngle(sumX, sumY);
	}
  }

  // assign orientation of the dominant response vector
  ip.orientation = (float)orientation;
}

__device__ float d_Gaussian(int x, int y, float sig)
{
  float pi = (float)M_PI;
  return ((float)1 / ((float)2 * pi * sig * sig)) * (float)exp(-(x * x + y * y) / ((float)2.0 * sig * sig));
}

__global__ void cudaGetDescriptor(float ** img, float * descriptor, float * len, int d_X, int d_Y, int d_S, float d_co, float d_si, int Height, int Width)
{
	extern __shared__ float floatdata[];

	int X = d_X;
	int Y = d_Y;
	int S = d_S;
	float co = d_co;
	float si = d_si;

	int ij_id[4] = {-12, -7, -2, 3};

	int i = ij_id[blockIdx.x];
	int j = ij_id[blockIdx.y];
	int bid = blockIdx.x*gridDim.x + blockIdx.y;
	float cx = (float)(-0.5) + (float)(1 + blockIdx.x);
	float cy = (float)(-0.5) + (float)(1 + blockIdx.y);

	int ix = i + 5;
	int jx = j + 5;

	float dx, dy, mdx, mdy;

	float xs = (int)floor(X + (-jx * S * si + ix * S * co) + (float)0.5);
	float ys = (int)floor(Y + (jx * S * co + ix * S * si) + (float)0.5);


	int k = i + threadIdx.x;
	int l = j + threadIdx.y;
	int height = Height;
	int width = Width;
	int idx = threadIdx.x*blockDim.x + threadIdx.y;

	//Get coords of sample point on the rotated axis
	int sample_x = (int)floor(X + (-l * S * si + k * S * co) + (float)0.5);
	int sample_y = (int)floor(Y + (l * S * co + k * S * si) + (float)0.5);

	//Get the gaussian weighted x and y responses
	float gauss_s1 = d_Gaussian(xs - sample_x, ys - sample_y, (float)2.5 * S);
	float rx = (float)d_HaarX(img, sample_y, sample_x, 2 * S, height, width);
	float ry = (float)d_HaarY(img, sample_y, sample_x, 2 * S, height, width);

	//Get the gaussian weighted x and y responses on rotated axis
	float rrx = gauss_s1 * (-rx * si + ry * co);
	float rry = gauss_s1 * (rx * co + ry * si);

	floatdata[0*81+idx] = rrx;
	floatdata[1*81+idx] = rry;
	floatdata[2*81+idx] = fabs(rrx);
	floatdata[3*81+idx] = fabs(rry);

	__syncthreads();

	if (idx == 39)
	{
		floatdata[0*81+39] += floatdata[0*81+80] + floatdata[0*81+79];
		floatdata[1*81+39] += floatdata[1*81+80] + floatdata[1*81+79];
		floatdata[2*81+39] += floatdata[2*81+80] + floatdata[2*81+79];
		floatdata[3*81+39] += floatdata[3*81+80] + floatdata[3*81+79];
	}

	else if (idx < 39)
	{
		floatdata[0*81+idx] += floatdata[0*81+idx + 40];
		floatdata[1*81+idx] += floatdata[1*81+idx + 40];
		floatdata[2*81+idx] += floatdata[2*81+idx + 40];
		floatdata[3*81+idx] += floatdata[3*81+idx + 40];
	}

	__syncthreads();

	if (idx < 20)
	{
		floatdata[0*81+idx] += floatdata[0*81+idx + 20];
		floatdata[1*81+idx] += floatdata[1*81+idx + 20];
		floatdata[2*81+idx] += floatdata[2*81+idx + 20];
		floatdata[3*81+idx] += floatdata[3*81+idx + 20];
	}

	__syncthreads();

	if (idx < 10)
	{
		floatdata[0*81+idx] += floatdata[0*81+idx + 10];
		floatdata[1*81+idx] += floatdata[1*81+idx + 10];
		floatdata[2*81+idx] += floatdata[2*81+idx + 10];
		floatdata[3*81+idx] += floatdata[3*81+idx + 10];
	}

	__syncthreads();

	if (idx < 5)
	{
		floatdata[0*81+idx] += floatdata[0*81+idx + 5];
		floatdata[1*81+idx] += floatdata[1*81+idx + 5];
		floatdata[2*81+idx] += floatdata[2*81+idx + 5];
		floatdata[3*81+idx] += floatdata[3*81+idx + 5];
	}

	__syncthreads();

	if (idx == 0)
	{
		floatdata[0*81+0] += floatdata[0*81+1] + floatdata[0*81+2] + floatdata[0*81+3] + floatdata[0*81+4];
		floatdata[1*81+0] += floatdata[1*81+1] + floatdata[1*81+2] + floatdata[1*81+3] + floatdata[1*81+4];
		floatdata[2*81+0] += floatdata[2*81+1] + floatdata[2*81+2] + floatdata[2*81+3] + floatdata[2*81+4];
		floatdata[3*81+0] += floatdata[3*81+1] + floatdata[3*81+2] + floatdata[3*81+3] + floatdata[3*81+4];

		dx =  floatdata[0*81+0];
		dy =  floatdata[1*81+0];
		mdx = floatdata[2*81+0];
		mdy = floatdata[3*81+0];
		//Add the values to the descriptor vector
		float gauss_s2 = d_Gaussian(cx - (float)2, cy - (float)2, (float)1.5);

		descriptor[0+bid] =  dx * gauss_s2;
		descriptor[1+bid] =  dy * gauss_s2;
		descriptor[2+bid] = mdx * gauss_s2;
		descriptor[3+bid] = mdy * gauss_s2;
		len[bid] = (dx * dx + dy * dy + mdx * mdx + mdy * mdy) * gauss_s2 * gauss_s2;
	}
}


/// <summary>
    /// Construct descriptor vector for this interest point
    /// </summary>
    /// <param name="bUpright"></param>
void SurfDescriptor::GetDescriptor(IPoint &ip, bool bUpright, bool bExtended, float * d_descriptor, float * d_len)
{
    float co, si;
    float len = (float)0;

    // Get rounded InterestPoint data
    int X = (int)floor(ip.x + (float)0.5);
    int Y = (int)floor(ip.y + (float)0.5);
    int S = (int)floor(ip.scale + (float)0.5);

    // Allocate descriptor memory
    ip.SetDescriptorLength(64);

    if (bUpright)
    {
		co = 1;
		si = 0;
    }
    else
    {
		co = (float)cos(ip.orientation);
		si = (float)sin(ip.orientation);
    }


	dim3 dimGrid(4, 4);
	dim3 dimBlock(9, 9);


	cudaGetDescriptor <<<dimGrid, dimBlock, 4*81*sizeof(float)>>> (img->Matrix, d_descriptor, d_len, X, Y, S, co, si, img->Height, img->Width);
	hipDeviceSynchronize();


	float h_len[16];
	hipMemcpy(ip.descriptor, d_descriptor, 64*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(        h_len,        d_len, 16*sizeof(float), hipMemcpyDeviceToHost);


	for (int ii = 0; ii < 16; ii++)
		len += h_len[ii];

	len = (float)sqrt((double)len);
	if (len > 0)
	{
		for (int d = 0; d < ip.descriptorLength; ++d)
		{
			ip.descriptor[d] /= len;
		}
	}
}


/// <summary>
/// Get the angle formed by the vector [x,y]
/// </summary>
/// <param name="X"></param>
/// <param name="Y"></param>
/// <returns></returns>
double SurfDescriptor::GetAngle(float X, float Y)
{
  if (X >= 0 && Y >= 0)
	return atan(Y / X);
  else if (X < 0 && Y >= 0)
	return M_PI - atan(-Y / X);
  else if (X < 0 && Y < 0)
	return M_PI + atan(Y / X);
  else if (X >= 0 && Y < 0)
	return 2 * M_PI - atan(-Y / X);

  return 0;
}


/// <summary>
/// Get the value of the gaussian with std dev sigma
/// at the point (x,y)
/// </summary>
/// <param name="x"></param>
/// <param name="y"></param>
/// <param name="sig"></param>
/// <returns></returns>
float SurfDescriptor::Gaussian(int x, int y, float sig)
{
  float pi = (float)M_PI;
  return ((float)1 / ((float)2 * pi * sig * sig)) * (float)exp(-(x * x + y * y) / ((float)2.0 * sig * sig));
}


/// <summary>
/// Get the value of the gaussian with std dev sigma
/// at the point (x,y)
/// </summary>
/// <param name="x"></param>
/// <param name="y"></param>
/// <param name="sig"></param>
/// <returns></returns>
float SurfDescriptor::Gaussian(float x, float y, float sig)
{
  float pi = (float)M_PI;
  return (float)1 / ((float)2 * pi * sig * sig) * (float)exp(-(x * x + y * y) / ((float)2.0 * sig * sig));
}

